#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "errors.h"
#include <algorithm>

using namespace std;

DeviceBool::DeviceBool(bool initial) {
    HANDLE_ERROR(hipMalloc(&device_data, sizeof(bool)));
    HANDLE_ERROR(hipHostMalloc(&host_data, sizeof(bool)));
    set_value(initial); // default value
}

DeviceBool::DeviceBool() : DeviceBool(false) {}

DeviceBool::~DeviceBool() {
    HANDLE_ERROR(hipHostFree(host_data));
    HANDLE_ERROR(hipFree(device_data));
    device_data = NULL;
}

void DeviceBool::set_value(bool val) {
    HANDLE_ERROR(hipMemset(device_data, val, sizeof(bool)));
}

bool DeviceBool::get_value() {
    HANDLE_ERROR(hipMemcpy(host_data, device_data, sizeof(bool), hipMemcpyDeviceToHost));
    return *host_data;
}

uint32_t num_verts(const vector<pair<uint32_t, uint32_t>> &edges) {
    // Compute number of vertices based on maximum vertex label.
    uint32_t n = 0;
    for (const auto &edge : edges) n = max({n, edge.first, edge.second});
    n++;
    return n;
}

// Helper function which converts graph into adjacency lists representation.
static vector<vector<uint32_t>> adjacency_lists(const vector<pair<uint32_t, uint32_t>> &edges) {
    uint32_t n = num_verts(edges);
    vector<vector<uint32_t>> adjs(n);

    for (auto edge : edges) {
        adjs[edge.first].push_back(edge.second);
        adjs[edge.second].push_back(edge.first);
    }

    return adjs;
}

VirtualCSR::VirtualCSR(const vector<pair<uint32_t, uint32_t>> &edges, uint32_t mdeg) {
    auto graph = adjacency_lists(edges);
        
    for (uint32_t v = 0; v < graph.size(); v++) { // iterate over real verts
        uint32_t u = 0; // index of adjacent vert in v's adjacency list
        while (u < graph[v].size()) {
            vmap.push_back(v); // map new virtual vert to real vert v
            vptrs.push_back(adjs.size()); // mark the beginning of virtual vert's adjacency list
            for (uint32_t deg = 0; deg < mdeg && u < graph[v].size(); deg++, u++) {
                adjs.push_back(graph[v][u]);
            }
        }
    }

    vptrs.push_back(adjs.size()); // add guard at the end of vptrs
}

// Returns grid_size based on the overall required number of threads and block size.
uint32_t grid_size(uint32_t min_threads_count, uint32_t block_size) {
    return (min_threads_count + block_size - 1) / block_size;
}

