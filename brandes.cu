
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <queue>
#include <stack>

using namespace std;

static void print_usage() {
    cout << "Program usage: ./brandes input-file output-file" << endl;
}

static void check_args(int argc, char *argv[]) {
    if (argc != 3 || !strcmp(argv[1], "") || !strcmp(argv[2], "")) {
        print_usage();
        exit(1);
    }
}

static vector<pair<int, int>> load_edges(string path) {
    ifstream ifs;
    ifs.open(path);
    if(!ifs.is_open()) {
        cerr << "Cannot open file \"" << path << "\"." << endl;
        exit(1);
    }

    vector<pair<int, int>> edges;

    int u, v;
    while(ifs >> u >> v) edges.emplace_back(u, v);

    ifs.close();
    return edges;
} 

static vector<float> compute_betweeness(const vector<pair<int, int>> &edges) {
    // Compute number of vertices based on maximum vertex label.
    int n = 0;
    for (const auto &edge : edges) n = max({n, edge.first, edge.second});
    n++;

    // Create graph as adjacency list, based on edges.
    vector<vector<int>> graph(n);
    for (const auto &edge : edges) {
        graph[edge.first].push_back(edge.second);
        graph[edge.second].push_back(edge.first);
    }

    // Based on https://kops.uni-konstanz.de/bitstream/handle/123456789/5739/algorithm.pdf
    // Compute centralities of each vertex.
    vector<float> centrality(n);
    for (int s = 0; s < n; s++) {
        queue<int> que; // BFS queue
        stack<int> stk; // verts ordered by distance from source
        vector<vector<int>> preds(n); // list of predecessors of each vertex
        vector<int> num_paths(n); // number of paths from source to each vertex
        vector<int> dist(n); // distance from source to each vertex
        num_paths[s] = 1; // there's 1 path to oneself
        fill(dist.begin(), dist.end(), -1); // allows dist to act as 'visited' marker
        dist[s] = 0; // dist to oneself is 0

        que.push(s);
        while(!que.empty()) {
            auto v = que.front();
            que.pop();
            stk.push(v);

            for (auto w : graph[v]) {
                if (dist[w] < 0) { // w visited for the first time
                    que.push(w);
                    dist[w] = dist[v] + 1;
                }
                if (dist[w] == dist[v] + 1) { // shortest path from source to w
                    num_paths[w] += num_paths[v];
                    preds[w].push_back(v);
                }
            }
        }

        vector<float> dependency(n);
        while (!stk.empty()) {
            auto w = stk.top();
            stk.pop();
            for (auto v : preds[w]) {
                dependency[v] += static_cast<float>(num_paths[v]) / num_paths[w] * (1.0f + dependency[w]);
            }
            if (w != s) {
                centrality[w] += dependency[w];
            }
        }
    }

    return centrality;
}

static void save_to_file(string path, const vector<float> &centrality) {
    ofstream ofs;
    ofs.open(path);

    if (!ofs.is_open()) {
        cerr << "Cannot open file \"" << path << "\"." << endl;
        exit(1);
    }

    for (auto val : centrality) {
        ofs << val << endl;
    }

    ofs.close();
}

int main(int argc, char *argv[]) {
    check_args(argc, argv);
    auto edges = load_edges(argv[1]);
    auto betweeness = compute_betweeness(edges);
    save_to_file(argv[2], betweeness);

    return 0;
}