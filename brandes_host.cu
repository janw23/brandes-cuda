#include "hip/hip_runtime.h"
#include "brandes_host.cuh"

#include <algorithm>

using namespace std;

CUDATimer::Timer::Timer(hipEvent_t start, hipEvent_t stop) : stop{stop} {
    hipEventRecord(start);
}

CUDATimer::Timer::~Timer() {
    hipEventRecord(stop);
}

CUDATimer::Timer CUDATimer::kernel_timer() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    kernel_evts.emplace_back(start, stop);
    return Timer(start, stop);
}


float CUDATimer::elapsed_time_kernels() {
    float time = 0;
    for (auto evt : kernel_evts) {
        hipEventSynchronize(evt.second);
        float ms;
        hipEventElapsedTime(&ms, evt.first, evt.second);
        time += ms;
    }
    return time;
}

float CUDATimer::elapsed_time_memcpy() {
    float time = 0;
    for (auto evt : memcpy_evts) {
        hipEventSynchronize(evt.second);
        float ms;
        hipEventElapsedTime(&ms, evt.first, evt.second);
        time += ms;
    }
    return time;
}

CUDATimer::~CUDATimer() {
    for (auto evt : kernel_evts) {
        hipEventDestroy(evt.first);
        hipEventDestroy(evt.second);
    }
}


int num_verts(const vector<pair<int, int>> &edges) {
    // Compute number of vertices based on maximum vertex label.
    int n = 0;
    for (const auto &edge : edges) n = max({n, edge.first, edge.second});
    n++;
    return n;
}

// Helper function which converts graph into adjacency lists representation.
static vector<vector<int>> adjacency_lists(const vector<pair<int, int>> &edges) {
    int n = num_verts(edges);
    vector<vector<int>> adjs(n);

    for (auto edge : edges) {
        adjs[edge.first].push_back(edge.second);
        adjs[edge.second].push_back(edge.first);
    }

    return adjs;
}

host::VirtualCSR::VirtualCSR(const vector<pair<int, int>> &edges, int mdeg) {
    auto graph = adjacency_lists(edges);
        
    for (int v = 0; v < graph.size(); v++) { // iterate over real verts
        int u = 0; // index of adjacent vert in v's adjacency list
        while (u < graph[v].size()) {
            vmap.push_back(v); // map new virtual vert to real vert v
            vptrs.push_back(adjs.size()); // mark the beginning of virtual vert's adjacency list
            for (int deg = 0; deg < mdeg && u < graph[v].size(); deg++, u++) {
                adjs.push_back(graph[v][u]);
            }
        }
    }

    vptrs.push_back(adjs.size()); // add guard at the end of vptrs
}
