#include "hip/hip_runtime.h"
#include "brandes_device.cuh"
#include "errors.h"

#include <iostream> // TODO remove
#include <cstdio> // TODO remove

device::VirtualCSR::VirtualCSR(const host::VirtualCSR &vcsr) {
    HANDLE_ERROR(hipMalloc(&vmap, sizeof(*vmap) * vcsr.vmap.size()));
    HANDLE_ERROR(hipMalloc(&vptrs, sizeof(*vptrs) * vcsr.vptrs.size()));
    HANDLE_ERROR(hipMalloc(&adjs, sizeof(*adjs) * vcsr.adjs.size()));

    HANDLE_ERROR(hipMemcpy(vmap, vcsr.vmap.data(), sizeof(*vmap) * vcsr.vmap.size(), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(vptrs, vcsr.vptrs.data(), sizeof(*vptrs) * vcsr.vptrs.size(), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(adjs, vcsr.adjs.data(), sizeof(*adjs) * vcsr.adjs.size(), hipMemcpyHostToDevice));
}

void device::VirtualCSR::free() {
    HANDLE_ERROR(hipFree(vmap));
    vmap = NULL;
    HANDLE_ERROR(hipFree(vptrs));
    vptrs = NULL;
    HANDLE_ERROR(hipFree(adjs));
    adjs = NULL;
}

__global__
void bc_virtual_forward(device::VirtualCSR vcsr, int layer, bool *cont) {
    printf("Hai CUDA\n");
    for (int i = 0; i < 3; i++) {
        printf("%d\n", vcsr.vmap[i]);
    }
}